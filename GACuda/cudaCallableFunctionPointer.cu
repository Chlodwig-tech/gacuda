#ifndef CUDA_CALLABLE_FUNCTION_POINTER
#define CUDA_CALLABLE_FUNCTION_POINTER

/*-----------------cudaCallableFunctionPointer-----------------*/

template<typename T> struct cudaCallableFunctionPointer{
  cudaCallableFunctionPointer(T* f_);
  ~cudaCallableFunctionPointer();
  T* ptr;
};

template<typename T> cudaCallableFunctionPointer<T>::cudaCallableFunctionPointer(T* f_){
    T* host_ptr = (T*)malloc(sizeof(T));
    CUDA_CALL(hipMalloc((void**)&ptr, sizeof(T)), "hipMalloc f_ (cudaCallableFunctionPointer)");

    CUDA_CALL(hipMemcpyFromSymbol(host_ptr, HIP_SYMBOL(*f_), sizeof(T)), "hipMemcpyFromSymbol f_ (cudaCallableFunctionPointer)");
    CUDA_CALL(hipMemcpy(ptr, host_ptr, sizeof(T), hipMemcpyHostToDevice), "hipMemcpy f_ (cudaCallableFunctionPointer)");
    
    free(host_ptr);
}

template<typename T> cudaCallableFunctionPointer<T>::~cudaCallableFunctionPointer(){
    CUDA_CALL(hipFree(ptr), "hipFree f_ (cudaCallableFunctionPointer)");
}

#endif // CUDA_CALLABLE_FUNCTION_POINTER