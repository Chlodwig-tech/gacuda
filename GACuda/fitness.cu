#include "hip/hip_runtime.h"
#ifndef FITNESS_CU
#define FITNESS_CU

#include "fitnessData.cu"
#include "cudaCallableFunctionPointer.cu"

#define CUDA_CALL(x, message) {if((x) != hipSuccess) { \
    printf("%s(%d): error: %s\n", __FILE__, __LINE__, message); \
    exit(EXIT_FAILURE); }}

/*--------------------------------Fitness--------------------------------*/

template<typename T> using FitnessFunc = T (*)(T*, int, T*);
template<typename T> __device__ T fitness(T *population, int index, T *FitnessData=nullptr);
template<typename T> __device__ FitnessFunc<T> fitness_func = fitness;

template<typename T> __global__ void fitness_kernel(
    T *fitnesses, T *population, int population_size, T*addata, FitnessFunc<T>* f)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    if(index < population_size){
        fitnesses[index] = (*f)(population, index, addata);
    }
}

template<typename T> class Fitness{
public:
    T *data;
    int size;
    FitnessData<T> *fitnessData;
public:
    Fitness(int size);
    ~Fitness();
    void uploadFitnessData(T *hdata, int size);
    void evaluate(T* population);
    void evaluate(T* population, T *hdata);
};

template<typename T> Fitness<T>::Fitness(int size) : size(size){
    CUDA_CALL(hipMalloc((void **)&data, size * sizeof(T)), "Fitness hipMalloc");
    fitnessData = nullptr;
}

template<typename T> Fitness<T>::~Fitness(){
    CUDA_CALL(hipFree(data), "Fitness hipFree");
    if(fitnessData != nullptr)
        delete fitnessData;
}

template<typename T> void Fitness<T>::uploadFitnessData(T *hdata, int size){
    fitnessData = new FitnessData<T>(hdata, size);
}

template<typename T> void Fitness<T>::evaluate(T* population){
    int numThreads = 1024;
    int numBlocks = size / numThreads + 1;
    cudaCallableFunctionPointer<FitnessFunc<T>> ccfp(&fitness_func<T>);
    fitness_kernel<<<numBlocks, numThreads>>>(data, population, size, fitnessData->data, ccfp.ptr);
    hipDeviceSynchronize();
}

template<typename T> void Fitness<T>::evaluate(T* population, T *hdata){
    evaluate(population);
    CUDA_CALL(hipMemcpy(hdata, data, size * sizeof(T), hipMemcpyDeviceToHost), "Fitness hipMemcpy d2h");
}

#endif // FITNESS_CU