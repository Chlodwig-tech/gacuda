#ifndef POPULATION_CU
#define POPULATION_CU

#define CUDA_CALL(x, message) {if((x) != hipSuccess) { \
    printf("%s(%d): error: %s\n", __FILE__, __LINE__, message); \
    exit(EXIT_FAILURE); }}
#define CUDA_GPU_CALL(x, message) {if((x) != hipSuccess) { \
    printf("%s(%d): error: %s\n", __FILE__, __LINE__, message);}}

template<typename T> __device__ void swap(T *a, T *b){
    T temp = *a;
    *a = *b;
    *b = temp;
}

template<typename T> __device__ void swap(T* a, int i, int j, int size){
    int n = size * sizeof(T);
    i *= size;
    j *= size;
    T *temp = (T*)malloc(n);
    memcpy(temp, &a[i], n);
    memcpy(&a[i], &a[j], n);
    memcpy(&a[j], temp, n);
    free(temp);
}

template<typename T> __global__ void bitonicSortKernel(T *array, int population_size, int individual_size, T *keys, int j, int k){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int ixj = tid ^ j;

    if(tid < population_size && ixj > tid && !(((tid & k) == 0) ^ (keys[tid] > keys[ixj]))){
        swap(array, tid, ixj, individual_size);
        swap(&keys[tid], &keys[ixj]);
    }
}

template<typename T> void bitonicSort(T *array, int population_size, int individual_size, T *keys){
    T *xd = new T[5];
    CUDA_CALL(hipMemcpy(xd, keys, 5 * sizeof(T), hipMemcpyDeviceToHost), "Fitness hipMemcpy d2h");
    for(int i = 0; i < 5; i++){
        printf("%d\n", xd[i]);
    }
    delete[] xd;
    int numThreads = 1024;
    int numBlocks = population_size / numThreads + 1;
    for(int k = 2; k <= population_size; k *= 2){
        for(int j = k >> 1; j > 0; j >>= 1){
            bitonicSortKernel<<<numBlocks, numThreads>>>(array, population_size, individual_size, keys, j, k);
            hipDeviceSynchronize();
        }
    }
}
    
/*--------------------------------Population--------------------------------*/

template<typename T> class Population{
public:
    T *data;
    int population_size;
    int individual_size;
    int size;

    void sort(T *fitnesses);
public:
    Population(int individual_size, int population_size);
    ~Population();
    void Get(T *hdata);
    void Set(T *hdata);
};


template<typename T> void Population<T>::sort(T *fitnesses){
    bitonicSort<T>(data, population_size, individual_size, fitnesses);
}

template<typename T> Population<T>::Population(
    int individual_size, int population_size
):
    individual_size(individual_size),
    population_size(population_size)
{
    size = individual_size * population_size;
    CUDA_CALL(hipMalloc((void **)&data, size * sizeof(T)), "Population hipMalloc");
}

template<typename T> Population<T>::~Population(){
    CUDA_CALL(hipFree(data), "Population hipFree");
}

template<typename T> void Population<T>::Get(T *hdata){
    CUDA_CALL(hipMemcpy(hdata, data, size * sizeof(T), hipMemcpyDeviceToHost), "Population hipMemcpy d2h");
}
template<typename T> void Population<T>::Set(T *hdata){
    CUDA_CALL(hipMemcpy(data, hdata, size * sizeof(T), hipMemcpyHostToDevice), "Population hipMemcpy h2d");
}

#endif // POPULATION_CU