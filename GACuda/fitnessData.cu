#ifndef FITNESS_DATA_H
#define FITNESS_DATA_H

/*--------------------------------FitnessData--------------------------------*/

template<typename T> struct FitnessData{
    T *data;
    int size;

    FitnessData(T *hdata, int size);
    ~FitnessData();
};

template<typename T> FitnessData<T>::FitnessData(T *hdata, int size) : size(size){
    CUDA_CALL(hipMalloc((void **)&data, size * sizeof(T)), "FitnessData hipMalloc");
    CUDA_CALL(hipMemcpy(data, hdata, size * sizeof(T), hipMemcpyHostToDevice), "FitnessData hipMemcpy h2d");
}

template<typename T> FitnessData<T>::~FitnessData(){
    CUDA_CALL(hipFree(data), "FitnessData hipFree");
}

#endif // FITNESS_DATA_H